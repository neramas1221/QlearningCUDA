#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include "win-gettimeofday.h"

// number of states
#define NUMBER_OF_STATES 11
// number of actions
#define NUMBER_OF_ACTIONS 4
// number of trials the agent will proform
#define NUMBER_OF_TRIALS 50
// number of epiosdes the agent will proform in each trial
#define NUMBER_OF_EPISODES 200

/*				 ___________________
GRID WORLD USED | 6 | 7 | 8 | 9 | 10|
				| 3 |   | 4 |   | 5 |
				| 0 |   | 1 |   | 2 |
*/

///////////////////////////////////////////////
//				seedRandom  				 //
// seedRanom is only used to seed the random //
// generator at the start of the programe	 //
///////////////////////////////////////////////

void seedRandom()
{
	//seeds the number generator to the system clock
	srand(time(NULL));
}

///////////////////////////////////////////////
//				transfer Function			 //
// transfer function is used to get the		 //
// next state after the agent proformes its	 //
// action in its current state				 //
// state : current state the agent is in	 //
// action : the move that the agent is going //
// to make in this grid world				 //
///////////////////////////////////////////////

int transferFunction(int state, int action)
{
	// creates value to store new state
	int newState = 0;
	// sets new state to state meaning if the agent does not
	// move its new state = its old state
	newState = state;
	// switch based on the action
	switch (action)
	{
	case 0: 
		// checks if this action can be proformed in one of the states, if it 
		// can be proformed move the agent to its new state if not then dont
		// move the agent
		if (state == 0 || state == 1 || state == 2 || state == 3)
		{
		
			newState = state + 3;
			break;
		}
		else if (state == 4)
		{
			
			newState = state + 4;
			break;
		}
		else if (state == 5)
		{
			
			newState = state + 5; 
			break;
		}
	case 1:
		if (state == 6 || state == 7 || state == 8 || state == 9)
		{
			newState = state + 1;
			break;
		}
	case 2:
		if (state == 3 || state == 4 || state == 5 || state == 6)
		{
			newState = state - 3;
			break;
		}
		else if (state == 8)
		{
			newState = 4;
			break;
		}
		else if (state == 10)
		{
			newState = 5;
			break;
		}
	case 3:
		if (state == 10 || state == 7 || state == 8 || state == 9)
		{
			newState = state - 1;
			break;
		}
	}
	// returns new state
	return newState;
}

///////////////////////////////////////////////
//				generateQtable				 //
// generate Qtable is used to create an 11 x //
// 4 table of values between 0.1-0.01		 //
// qTable : 2-D array that is filled with the//
// Q-values which corresponds to the state	 //
// and action								 //
///////////////////////////////////////////////

void generateQtable(float **qTable)
{
	// loop over the number of states
	for (int i = 0; i < NUMBER_OF_STATES; i++)
	{ // loops over the number of actions
		for (int j = 0; j < NUMBER_OF_ACTIONS; j++)
		{
			// geneates a random double between 0.1 -0.01
			qTable[i][j] = (double)rand() / (double)((unsigned)RAND_MAX + 0.1)* (0.01);
		}
	}
}

///////////////////////////////////////////////
//				getAction					 //
// get Action will return an action that the //
// agent will proform in its state. This is  //
// tipically the highest Q-value for that	 //
// state									 //
// state : the state the agent is in		 //
// qTable : 2-D array that is filled with	 //
// the Q-Values of states and actions		 //
///////////////////////////////////////////////

int getAction(float **qTable, int state)
{
	int action;
	double maxValue = -1;
	// loops over the number of actions
	for (int i = 0; i < NUMBER_OF_ACTIONS; i++)
	{
		// finds the max value in the qTable
		if (qTable[state][i] >maxValue)
		{
			// sets the max value to this Q- value
			maxValue = qTable[state][i];
			// sets the action to the best action
				action = i;
		}
	}
	// returns the action
	return action;
}

///////////////////////////////////////////////
//				getAction					 //
// get Action will return an action that the //
// agent will proform in its state. This is  //
// tipically the highest Q-value for that	 //
// state how every there is a 1/10 chance	 //
// that a random action will be chosen for	 //
// the agent to proform						 //
// state : the state the agent is in		 //
// qTable : 2-D array that is filled with	 //
// the Q-Values of states and actions		 //
///////////////////////////////////////////////

int getRndAction(float **qTable, int state)
{
	int action = 0;
	double maxValue = -1;
	int rndValue;
	// gets a random value for the exploration rate
	rndValue = rand() % (10 + 1 - 1) + 1;
	// if its not exploring
	if (rndValue <= 9)
	{
		// loops over the action
		for (int i = 0; i < NUMBER_OF_ACTIONS; i++)
		{
			// checks the best value against the current Q-value
			if (qTable[state][i] > maxValue)
			{
				// sets the action to the best action
				action = i;
				// updates the max value
				maxValue = qTable[state][i];
			}
		}
	}
	// else explore
	else
	{
		// randomly generates an action
		rndValue = rand() % (3 + 1 - 0) + 0;
		// sets the action
		action = rndValue;
	}
	return action;
}

///////////////////////////////////////////////
//				getReward					 //
// get reward is used to give the agent its  //
// reward based on its state and action it	 //
// would take								 //
///////////////////////////////////////////////

int getReward(int state, int action)
{
	int reward = 0;
	// checks if the action and state will
	// lead to the goal state being reach
	if (state == 4 && action == 2)
	{
		// set reward to 10
		reward = 10;
	}

	// return the reward
	return reward;
}

///////////////////////////////////////////////
//				update Qtable   			 //
// update Q-Table is used to update the last //
// move the agent profomed in the grid world //
// using its reward, next state and currrent //
// state and action	and updates the Q value  //
// for that move							 //
// qTable : is an array that stores all of   //
// actions q - values for a given state		 //
// state : is the current state the agent is //
// in										 //
// action : the action the agent profomed    //
// in its given state						 //
// newState : is the state the agent ended	 //
// up in based on its state and action		 //
// that the agent proformed					 //
// reward : reward that the agent earns		 //
// based on its state and action			 //
///////////////////////////////////////////////

double updateQTable(int state, int action, int nextState,int reward, float** qTable )
{
	double updateValue = 0;
	int nextAction;
	// discounts the learning by 0.9
	double discount = 0.9;
	double learningRate =0.2;
	// used to store the diffrence in the Q-table values
	double diff;
	// gets the best action that is mostlikely going to be proformed
	// in the agents next state
	nextAction = getAction(qTable, nextState);
	// calculates how good the move is
	diff = reward + discount * qTable[nextState][nextAction] - qTable[state][action];
	// work out the new Q values using the learning rate and the diffrence
	updateValue = qTable[state][action] + learningRate * diff;
	return updateValue;
}

///////////////////////////////////////////////
//				rndState		   			 //
// rnd state is used to randomlly generate a //
// starting state that is not the goal state //
///////////////////////////////////////////////

int rndState()
{
	int state = 0;
	int Max = 10;
	// calculates a random number betwen 0 - 10
	state = rand() % Max;
	// checks the state is not the goal state
	while (state == 1)
	{
		// generates a new state if it is
		state = rand() % Max;
	}
	
	return state;
}

///////////////////////////////////////////////
//				calculateSTD	  			 //
// calculate STD is used to calculates the	 //
// standard deviation on each trial the		 //
// agent has proformed, this method is only	 //
// run on the device						 //
// episodeCount : stores all of the episodes //
// that the agent proformed in this trial	 //
// total steps is the total number of steps	 //
// for that trial							 //
// size  : is the number of episodes the	 //
// agent completed							 //
///////////////////////////////////////////////

__device__ float calculateSTD(int episodeCounter[],int stepTotal,int size)
{
	float std;
	// gets the mean of the data
	float mean = stepTotal / size;
	// loops over the data
	for (int i = 0; i < size; i++)
	{
		// calculates the total of the data set squared
		std += pow(episodeCounter[i] - mean, 2);
	}
	// square roots the data devided by the size
	std = sqrt(std / size);
	return std;
}

///////////////////////////////////////////////
//				calculateAllSteps  			 //
// calculate all steps: is used to sum all	 //
// steps and there standard deviation per	 //
// trial									 //
///////////////////////////////////////////////

__global__ void calculateAllSteps(int *StepsArray)
{
	// creates arrrys to store the total number of steps in
	// the each trial and the steps in all episodes
	 int episodeStorage[NUMBER_OF_EPISODES];
	 int trialStorage[NUMBER_OF_TRIALS];
 	__shared__ int total;
	__shared__ int *allSteps;
	// sets the array the kernal is passed to one in shared memory
	allSteps = StepsArray;
	__shared__ int episodeAvrage;
	// creates array to store the standard deviation
	float std[NUMBER_OF_TRIALS];
	__shared__ int worstSteps;
	__shared__ int bestSteps;
	__shared__ int totalSteps;
	 bestSteps = 100000;
	 worstSteps = 0;
	total = 0; 
	episodeAvrage = 0;
	totalSteps = 0;
	// syncthreads to make sure all data has been created
	__syncthreads();
	// loops over the number of trials
	for (int i = 0; i < NUMBER_OF_TRIALS;i++)
	{
		// loops over the number of episodes
		for (int j = 0; j < NUMBER_OF_EPISODES; j+=20)
		{
			/* gets the element from the array using the stride values for example if i is 2 and j is 1 then the index
			 would be 201 meaning it is the second episode of the second trial;*/
			/*
			 a number of diffrent unravling amounts where used here and 20 seemed to yeld the best
			 results in terms of proformance and computation time.
			*/
			total += allSteps[i*NUMBER_OF_EPISODES + j];
			total += allSteps[i*NUMBER_OF_EPISODES + j+1];
			total += allSteps[i*NUMBER_OF_EPISODES + j+2];
			total += allSteps[i*NUMBER_OF_EPISODES + j+3];
			total += allSteps[i*NUMBER_OF_EPISODES + j+4];
			total += allSteps[i*NUMBER_OF_EPISODES + j+5];
			total += allSteps[i*NUMBER_OF_EPISODES + j+6];
			total += allSteps[i*NUMBER_OF_EPISODES + j+7];
			total += allSteps[i*NUMBER_OF_EPISODES + j+8];
			total += allSteps[i*NUMBER_OF_EPISODES + j+9];
			total += allSteps[i*NUMBER_OF_EPISODES + j+10];
			total += allSteps[i*NUMBER_OF_EPISODES + j+11];
			total += allSteps[i*NUMBER_OF_EPISODES + j+12];
			total += allSteps[i*NUMBER_OF_EPISODES + j+13];
			total += allSteps[i*NUMBER_OF_EPISODES + j+14];
			total += allSteps[i*NUMBER_OF_EPISODES + j+15];
			total += allSteps[i*NUMBER_OF_EPISODES + j+16];
			total += allSteps[i*NUMBER_OF_EPISODES + j+17];
			total += allSteps[i*NUMBER_OF_EPISODES + j+18];
			total += allSteps[i*NUMBER_OF_EPISODES + j+19];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+1];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+2];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+3];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+4];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+5];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+6];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+7];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+8];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+9];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+10];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+11];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+12];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+13];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+14];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+15];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+16];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+17];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+18];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+19];
		}
		// calculates the STD of the episode
		std[i] = calculateSTD(episodeStorage,total,NUMBER_OF_EPISODES);
		// stores the total number of steps accross this episode
		trialStorage[i] = total;
		// adds to the total steps across all trials
		totalSteps += total;
		// works out the avrage number of steps for that episode
		episodeAvrage += (total / NUMBER_OF_EPISODES);
		total = 0;
	}
	// calculates the avrage standard deviation
	float aveStd = 0;
	float mean = totalSteps / NUMBER_OF_TRIALS;
	for (int i = 0; i < NUMBER_OF_TRIALS; i++)
	{
		aveStd += pow(std[i] - mean, 2);
	}
	aveStd = sqrt(aveStd / NUMBER_OF_TRIALS);
	// loops over the data set
	for (int i = 0; i < NUMBER_OF_EPISODES*NUMBER_OF_TRIALS; i++)
	{
		// finds the best and worst number of steps from all of the
		// episodes the agent proformed
		if (bestSteps > allSteps[i])
		{
			bestSteps = allSteps[i];
		}
		if (worstSteps < allSteps[i])
		{
			worstSteps = allSteps[i];
		}
		__syncthreads();
	}
	// makes sure all of the threads have finished
	__syncthreads();
	// works out the threads block id
	int block_id = blockIdx.x + gridDim.x * blockIdx.y;
	// works out the threads id
	unsigned int id = blockDim.x * block_id + threadIdx.x;
	if (id == 1)
	{
		// thread one prints all of the data to the user so that no time is spent 
		// tranfering data back to the host when it is not going to be used
		printf("Data from trials \n\n");
		printf("avarge Standard deviation : %f\n", aveStd);
		printf("Best value %d\n", bestSteps);
		printf("worst value %d\n", worstSteps);
		printf("Avarage steps per tiral : %d\n",episodeAvrage);
		printf("Avarage steps per episode : %d\n", episodeAvrage/50);
		printf("\n\n");
	}
}
int main()
{
	// starts the programme timmer
	long long totalProgrammeTimmer = start_timer();
	// seeds the random number generator
	seedRandom();
	// sets the size of the arrys
	int size = sizeof(int) * NUMBER_OF_EPISODES * NUMBER_OF_TRIALS;
	// creates the qTable
	float **qTable;
	// allocates space to the q-table 
	qTable = (float**)  malloc(NUMBER_OF_STATES * sizeof(float*));
	// loops over the q-table and allocates space to each row equal to the number
	// of actions
	for (int i = 0; i < NUMBER_OF_STATES; i++)
	{
		qTable[i] = (float*)malloc(sizeof(float)*NUMBER_OF_ACTIONS);
	}
	// generates the q-table
	generateQtable(qTable);
		// creates array to store all steps across trials
	int *allSteps;
	// allocates space to the array 
	allSteps = (int*)malloc(size);
	// creates the data to store the values for the
	// q-learning algorithum
	int state = 0;
	int action = 0;
	int newState = 0;
	int reward = 0;
	int steps = 0;
	float updatedValue = 0;
	// loops over the number of trials
	for (int trial = 0; trial < NUMBER_OF_TRIALS; trial++)
	{
		// loops over the number of episodes
		for (int i = 0; i < NUMBER_OF_EPISODES; i++)
		{
			// randomly generates a starting state
			state = rndState();
			// loops until goal state is reached
			while (state != 1)
			{
				// gets the action for the state
				action = getRndAction(qTable, state);
				// gets the agents reward for the move
				reward = getReward(state, action);
				// gets the resulting state
				newState = transferFunction(state, action);
				// gets the new Q-value for that state and action
				updatedValue = updateQTable(state, action, newState, reward, qTable);
				// sets q-table at the state and action to be the new value
				qTable[state][action] = updatedValue;
				// updates the state
				state = newState;
				// updates the step count
				steps++;
			}
			// updates the array to have the total steps for that episode
			allSteps[trial*NUMBER_OF_EPISODES + i] = steps;
			// resest the steps
			steps = 0;
			
		}
		// generates a new qTable after each trial
		generateQtable(qTable);
	}
	// creates an aray to store all steps on the device
	int *d_allsteps;
	// startes the transfer timmer
	long long transferTimmer = start_timer();

	// creates error variable
	hipError_t err;
	// mallocs space for the data on the device
	err = hipMalloc((void**)&d_allsteps, size);
	
	printf("CUDA malloc 1D array: %s\n", hipGetErrorString(err));
	// copys the data over to the device
	err = hipMemcpy(d_allsteps, allSteps, size, hipMemcpyHostToDevice);
	printf("CUDA memcpy 1D array: %s\n", hipGetErrorString(err));
	// stops timmer
	stop_timer(transferTimmer,"Transfer timer");
	// starts computation timer
	long long GPUComput = start_timer();
	// calculates the grid size
	int gridSize = (int)ceil(NUMBER_OF_EPISODES*NUMBER_OF_TRIALS / NUMBER_OF_TRIALS);
	dim3 dimGrid(gridSize,1,1);
	// runs the kernal with the calculated block size and 50 threads
	calculateAllSteps << <gridSize, 50>> > (d_allsteps);
	// stops the timer
	stop_timer(GPUComput, "Computaion time");
	// stopes the total programme timmer
	stop_timer(totalProgrammeTimmer, "Total programme time");
	// frees memory
	hipFree(d_allsteps);

	free(allSteps);

	free(qTable);
}