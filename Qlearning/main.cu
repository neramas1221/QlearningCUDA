#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include "win-gettimeofday.h"


#define NUMBER_OF_STATES 11
#define NUMBER_OF_ACTIONS 4
#define NUMBER_OF_TRIALS 50
#define NUMBER_OF_EPISODES 200

void seedRandom()
{
	srand(time(NULL));
}
int transferFunction(int state, int action)
{
	int newState = 0;
	newState = state;

	switch (action)
	{
	case 0: 
		if (state == 0 || state == 1 || state == 2 || state == 3)
		{
			newState = state + 3;
			break;
		}
		else if (state == 4)
		{
			newState = state + 4;
			break;
		}
		else if (state == 5)
		{
			newState = state + 5; 
			break;
		}
	case 1:
		if (state == 6 || state == 7 || state == 8 || state == 9)
		{
			newState = state + 1;
			break;
		}
	case 2:
		if (state == 3 || state == 4 || state == 5 || state == 6)
		{
			newState = state - 3;
			break;
		}
		else if (state == 8)
		{
			newState = 4;
			break;
		}
		else if (state == 10)
		{
			newState = 5;
			break;
		}
	case 3:
		if (state == 10 || state == 7 || state == 8 || state == 9)
		{
			newState = state - 1;
			break;
		}
	}
	return newState;
}
void generateQtable(float **qTable)
{
	for (int i = 0; i < NUMBER_OF_STATES; i++)
	{
		for (int j = 0; j < NUMBER_OF_ACTIONS; j++)
		{
			qTable[i][j] = (double)rand() / (double)((unsigned)RAND_MAX + 1)* (0.1);
		}
	}
}
int getAction(float **qTable, int state)
{
	int action;
	double maxValue = -1;

	for (int i = 0; i < NUMBER_OF_ACTIONS; i++)
	{
		if (qTable[state][i] >maxValue)
		{
			maxValue = qTable[state][i];
				action = i;
		}
	}

	return action;
}
int getRndAction(float **qTable, int state)
{
	int action = 0;
	double maxValue = -1;
	int rndValue;
	rndValue = rand() % (10 + 1 - 1) + 1;
	if (rndValue <= 9)
	{
		for (int i = 0; i < NUMBER_OF_ACTIONS; i++)
		{
			if (qTable[state][i] > maxValue)
			{
				action = i;
				maxValue = qTable[state][i];
			}
		}
	}
	else
	{
		rndValue = rand() % (3 + 1 - 0) + 0;
		action = rndValue;
	}
	return action;
}
int getReward(int state, int action)
{
	int reward = 0;
	if (state == 4 && action == 2)
	{
		reward = 10;
	}
	return reward;
}
double updateQTable(int state, int action, int nextState,int reward, float** qTable )
{
	double updateValue = 0;
	int nextAction;
	double discount = 0.9;
	double learningRate =0.2;
	double diff;

	nextAction = getAction(qTable, nextState);
	diff = reward + discount * qTable[nextState][nextAction] - qTable[state][action];
	updateValue = qTable[state][action] + learningRate * diff;

	return updateValue;
}
int rndState()
{
	int state = 0;
	int Max = 10;
	state = rand() % Max;
	while (state == 1)
	{
		state = rand() % Max;
	}
	
	return state;
}
__device__ double calculateSTD(int episodeCounter[],int stepTotal)
{
	double std;
	double mean = stepTotal / NUMBER_OF_EPISODES;
	for (int i = 0; i < NUMBER_OF_EPISODES; i++)
	{
		std += pow(episodeCounter[i] - mean, 2);
	}
	std = sqrt(std / NUMBER_OF_EPISODES);
	return std;
}
__global__ void calculateAllSteps(int *StepsArray)
{
	__shared__ int episodeStorage[NUMBER_OF_EPISODES];
	__shared__ int trialStorage[NUMBER_OF_TRIALS];
 	__shared__ int total;
	__shared__ int *allSteps;
	allSteps = StepsArray;
	__shared__ int episodeTotal;
	//__shared__ double std[NUMBER_OF_TRIALS];
	__shared__ int worstSteps;
	__shared__ int bestSteps;
	 bestSteps = 100000;
	 worstSteps = 0;
	total = 0; 
	episodeTotal = 0;
	__syncthreads();
	for (int i = 0; i < NUMBER_OF_TRIALS;i++)
	{
		for (int j = 0; j < NUMBER_OF_EPISODES; j+=20)
		{
			/* gets the element from the array using the stride values for example if i is 2 and j is 1 then the index
			 would be 201 meaning it is the second episode of the second trial;*/
			total += allSteps[i*NUMBER_OF_EPISODES + j];
			total += allSteps[i*NUMBER_OF_EPISODES + j+1];
			total += allSteps[i*NUMBER_OF_EPISODES + j+2];
			total += allSteps[i*NUMBER_OF_EPISODES + j+3];
			total += allSteps[i*NUMBER_OF_EPISODES + j+4];
			total += allSteps[i*NUMBER_OF_EPISODES + j+5];
			total += allSteps[i*NUMBER_OF_EPISODES + j+6];
			total += allSteps[i*NUMBER_OF_EPISODES + j+7];
			total += allSteps[i*NUMBER_OF_EPISODES + j+8];
			total += allSteps[i*NUMBER_OF_EPISODES + j+9];
			total += allSteps[i*NUMBER_OF_EPISODES + j+10];
			total += allSteps[i*NUMBER_OF_EPISODES + j+11];
			total += allSteps[i*NUMBER_OF_EPISODES + j+12];
			total += allSteps[i*NUMBER_OF_EPISODES + j+13];
			total += allSteps[i*NUMBER_OF_EPISODES + j+14];
			total += allSteps[i*NUMBER_OF_EPISODES + j+15];
			total += allSteps[i*NUMBER_OF_EPISODES + j+16];
			total += allSteps[i*NUMBER_OF_EPISODES + j+17];
			total += allSteps[i*NUMBER_OF_EPISODES + j+18];
			total += allSteps[i*NUMBER_OF_EPISODES + j+19];
			//total += allSteps[i*NUMBER_OF_EPISODES + j+20];
			//total += allSteps[i*NUMBER_OF_EPISODES + j +21];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 22];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 23];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 24];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 25];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 26];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 27];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 28];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 29];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 30];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 31];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 32];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 33];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 34];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 35];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 36];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 37];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 38];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 39];
			//total += allSteps[i*NUMBER_OF_EPISODES + j+ 40];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 41];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 42];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 43];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 44];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 45];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 46];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 47];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 48];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 49];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 50];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 51];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 52];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 53];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 54];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 55];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 56];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 57];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 58];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 59];
			//total += allSteps[i*NUMBER_OF_EPISODES + j+ 60];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 61];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 62];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 63];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 64];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 65];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 66];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 67];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 68];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 69];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 70];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 71];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 72];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 73];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 74];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 75];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 76];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 77];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 78];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 79];
			//total += allSteps[i*NUMBER_OF_EPISODES + j+80];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 81];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 82];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 83];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 84];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 85];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 86];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 87];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 88];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 89];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 90];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 91];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 92];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 93];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 94];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 95];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 96];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 97];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 98];
			//total += allSteps[i*NUMBER_OF_EPISODES + j + 99];
			/*episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+1];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+2];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+3];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+4];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+5];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+6];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+7];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+8];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+9];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+10];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+11];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+12];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+13];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+14];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+15];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+16];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+17];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+18];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+19];*/
		
		}
		//std[i] = calculateSTD(episodeStorage,total);
		trialStorage[i] = total;
		episodeTotal += (total / 200);
		total = 0;
	}

	for (int i = 0; i < NUMBER_OF_EPISODES*NUMBER_OF_TRIALS; i++)
	{
		if (bestSteps > allSteps[i])
		{
			bestSteps = allSteps[i];
		}
		if (worstSteps < allSteps[i])
		{
			worstSteps = allSteps[i];
		}
		__syncthreads();
		//if (bestSteps > allSteps[i+1])
		//{
		//	bestSteps = allSteps[i+1];
		//}
		//if (worstSteps < allSteps[i+1])
		//{
		//	worstSteps = allSteps[i+1];
		//}
		//__syncthreads();
	}

	__syncthreads();
	int block_id = blockIdx.x + gridDim.x * blockIdx.y;
	unsigned int id = blockDim.x * block_id + threadIdx.x;
	//printf("id : %d\n", id);
	if (id == 1)
	{
		printf("Best value %d\n", bestSteps);
		printf("worst value %d\n", worstSteps);
		printf("Avarage steps per tiral : %d\n",episodeTotal);
		printf("Avarage steps per episode : %d\n", episodeTotal/50);
	}
	//returnData[0] = episodeTotal;
	//returnData[1] = episodeTotal / 50;
	//returnData[2] = bestSteps;
	//returnData[3] = worstSteps;
}
int main()
{
	long long totalProgrammeTimmer = start_timer();
	seedRandom();
	int size = sizeof(int) * NUMBER_OF_EPISODES * NUMBER_OF_TRIALS;
	int returnSize = sizeof(int) * 4;
	float **qTable;
	qTable = (float**)  malloc(NUMBER_OF_STATES * sizeof(float*));
	for (int i = 0; i < NUMBER_OF_STATES; i++)
	{
		qTable[i] = (float*)malloc(sizeof(float)*NUMBER_OF_ACTIONS);
	}
	generateQtable(qTable);
		
	int *allSteps;
	allSteps = (int*)malloc((NUMBER_OF_TRIALS * NUMBER_OF_EPISODES)* sizeof(int));
	/*allSteps[0] = (int*)malloc(NUMBER_OF_EPISODES * sizeof(int));
	for (size_t i = 1; i < NUMBER_OF_TRIALS; i++)
	{
		allSteps[i] = allSteps[i - 1] + NUMBER_OF_EPISODES;
	}*/
	/*for (int i = 0; i < NUMBER_OF_TRIALS; i++)
	{
		allSteps[i] = (int*)malloc(sizeof(int)* NUMBER_OF_EPISODES);
	}*/
	

	int state = 0;
	int action = 0;
	int newState = 0;
	int reward = 0;
	int steps = 0;
	float updatedValue = 0;

	for (int trial = 0; trial < NUMBER_OF_TRIALS; trial++)
	{
		for (int i = 0; i < NUMBER_OF_EPISODES; i++)
		{
			state = rndState();
			while (state != 1)
			{
				action = getRndAction(qTable, state);
				reward = getReward(state, action);
				newState = transferFunction(state, action);
				updatedValue = updateQTable(state, action, newState, reward, qTable);
				qTable[state][action] = updatedValue;
				
				state = newState;
				steps++;
			}
			
			allSteps[trial*NUMBER_OF_EPISODES + i] = steps;
			
			steps = 0;
			
		}
		generateQtable(qTable);
	}


	/*int *returnData;
	returnData =(int*) malloc(4 * sizeof(int));*/
	int *d_allsteps;
	//int *d_returnData;
	long long transferTimmer = start_timer();

	hipStream_t stream1;
	hipError_t err;
	err = hipStreamCreate(&stream1);

	printf("CUDA Stream: %s\n", hipGetErrorString(err));

	err = hipMalloc((void**)&d_allsteps, size);
	
	printf("CUDA malloc 1D array: %s\n", hipGetErrorString(err));

	//err = hipMalloc((void**)&d_returnData, returnSize);

	//printf("CUDA malloc 1D array: %s\n", hipGetErrorString(err));

	err = hipMemcpyAsync(d_allsteps, allSteps,size, hipMemcpyHostToDevice,stream1);

	printf("CUDA memcpy 1D array: %s\n", hipGetErrorString(err));

	//err = hipMemset(d_returnData, 0, returnSize); //(d_returnData,returnData,returnSize,hipMemcpyHostToDevice);

	//printf("CUDA memcpy 1D array of nothing: %s\n", hipGetErrorString(err));

	stop_timer(transferTimmer,"Transfer timer");

	long long GPUComput = start_timer();
	int gridSize = (int)ceil(NUMBER_OF_EPISODES*NUMBER_OF_TRIALS / 50);
	dim3 dimGrid(gridSize,1,1);
	dim3 dimBlock(50,1,1);

	calculateAllSteps << <gridSize, 50,0 ,stream1 >> > (d_allsteps);

	//err = hipMemcpy(returnData, d_returnData, returnSize, hipMemcpyDeviceToHost);

	//printf("CUDA memcpy data back: %s\n", hipGetErrorString(err));
	stop_timer(GPUComput, "Computaion time");
	/*printf("Best value %d\n", returnData[2]);
	printf("worst value %d\n", returnData[3]);
	printf("Avarage steps per tiral : %d\n", returnData[0]);
	printf("Avarage steps per episode : %d\n", returnData[1]);*/
	stop_timer(totalProgrammeTimmer, "Total programme time");

	hipFree(d_allsteps);

	//hipFree(d_returnData);

	free(allSteps);

	free(qTable);
}