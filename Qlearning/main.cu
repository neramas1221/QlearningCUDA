#include <stdio.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>


#include <time.h>

#define NUMBER_OF_STATES 11
#define NUMBER_OF_ACTIONS 4
#define NUMBER_OF_TRIALS 50
#define NUMBER_OF_EPISODES 200

void seedRandom()
{
	srand(time(NULL));
}
int transferFunction(int state, int action)
{
	int newState = 0;
	newState = state;

	switch (action)
	{
	case 0: 
		if (state == 0 || state == 1 || state == 2 || state == 3)
		{
			newState = state + 3;
			break;
		}
		else if (state == 4)
		{
			newState = state + 4;
			break;
		}
		else if (state == 5)
		{
			newState = state + 5; 
			break;
		}
	case 1:
		if (state == 6 || state == 7 || state == 8 || state == 9)
		{
			newState = state + 1;
			break;
		}
	case 2:
		if (state == 3 || state == 4 || state == 5 || state == 6)
		{
			newState = state - 3;
			break;
		}
		else if (state == 8)
		{
			newState = 4;
			break;
		}
		else if (state == 10)
		{
			newState = 5;
			break;
		}
	case 3:
		if (state == 10 || state == 7 || state == 8 || state == 9)
		{
			newState = state - 1;
			break;
		}
	}
	return newState;
}
void generateQtable(double **qTable)
{
	for (int i = 0; i < NUMBER_OF_STATES; i++)
	{
		for (int j = 0; j < NUMBER_OF_ACTIONS; j++)
		{
			qTable[i][j] = (double)rand() / (double)((unsigned)RAND_MAX + 1)* (0.1);
		}
	}
}
int getAction(double **qTable, int state)
{
	int action;
	double maxValue = -1;

	for (int i = 0; i < NUMBER_OF_ACTIONS; i++)
	{
		if (qTable[state][i] >maxValue)
		{
			maxValue = qTable[state][i];
				action = i;
		}
	}

	return action;
}
int getRndAction(double **qTable, int state)
{
	int action = 0;
	double maxValue = -1;
	int rndValue;
	rndValue = rand() % (10 + 1 - 1) + 1;
	if (rndValue <= 9)
	{
		for (int i = 0; i < NUMBER_OF_ACTIONS; i++)
		{
			if (qTable[state][i] > maxValue)
			{
				action = i;
				maxValue = qTable[state][i];
			}
		}
	}
	else
	{
		rndValue = rand() % (3 + 1 - 0) + 0;
		action = rndValue;
	}
	return action;
}
int getReward(int state, int action)
{
	int reward = 0;
	if (state == 4 && action == 2)
	{
		reward = 10;
	}
	return reward;
}
double updateQTable(int state, int action, int nextState,int reward,double** qTable )
{
	double updateValue = 0;
	int nextAction;
	double discount = 0.9;
	double learningRate =0.2;
	double diff;

	nextAction = getAction(qTable, nextState);
	diff = reward + discount * qTable[nextState][nextAction] - qTable[state][action];
	updateValue = qTable[state][action] + learningRate * diff;

	return updateValue;
}
int rndState()
{
	int state = 0;
	int Max = 10;
	state = rand() % Max;
	while (state == 1)
	{
		state = rand() % Max;
	}
	
	return state;
}
__device__ double calculateSTD(int episodeCounter[],int stepTotal)
{
	double std;
	double mean = stepTotal / NUMBER_OF_EPISODES;
	for (int i = 0; i < NUMBER_OF_EPISODES; i++)
	{
		std += pow(episodeCounter[i] - mean, 2);
	}
	std = sqrt(std / NUMBER_OF_EPISODES);
	return std;
}
__global__ void calculateAllSteps(int **StepsArray)
{
	__shared__ int epsiodeCounter[NUMBER_OF_EPISODES];
	__shared__ int trialCounter[NUMBER_OF_TRIALS];
	__shared__ int total;
	double std[NUMBER_OF_TRIALS];
	total = 0; 
	for (int i = 0; i < NUMBER_OF_TRIALS;i++)
	{
		for (int j = 0; j < NUMBER_OF_EPISODES; j++)
		{
			total += StepsArray[i][j];
			epsiodeCounter[j] = StepsArray[i][j];
		}
		std[i] = calculateSTD(epsiodeCounter,total);
		trialCounter[i] = total;
		total = 0;
	}
}
int main()
{
	seedRandom();

	double **qTable;
	qTable = (double**)  malloc(NUMBER_OF_STATES * sizeof(double*));
	for (int i = 0; i < NUMBER_OF_STATES; i++)
	{
		qTable[i] = (double*)malloc(sizeof(double)*NUMBER_OF_ACTIONS);
	}
	generateQtable(qTable);
		/*for (int i = 0; i < NUMBER_OF_STATES; i++)
		{
			for (int j = 0; j < NUMBER_OF_ACTIONS; j++)
			{
				printf("vlaue at [%d][%d] : %f\n", i, j, qTable[i][j]);
			}
		}*/
	int **allSteps;
	allSteps = (int**)malloc(NUMBER_OF_TRIALS * sizeof(int*));
	for (int i = 0; i < NUMBER_OF_TRIALS; i++)
	{
		allSteps[i] = (int*)malloc(sizeof(int)* NUMBER_OF_EPISODES);
	}

	int state = 0;
	int action = 0;
	int newState = 0;
	int reward = 0;
	int steps = 0;
	double updatedValue = 0;
	for (int trial = 0; trial < NUMBER_OF_TRIALS; trial++)
	{
		for (int i = 0; i < NUMBER_OF_EPISODES; i++)
		{
			state = rndState();
			while (state != 1)
			{
				action = getRndAction(qTable, state);
				reward = getReward(state, action);
				newState = transferFunction(state, action);
				updatedValue = updateQTable(state, action, newState, reward, qTable);
				qTable[state][action] = updatedValue;
				//printf("state : %d , action : %d newState : %d\n", state, action, newState);
				state = newState;
				steps++;
			}
			allSteps[trial][i] = steps;
			steps = 0;
			
		}
		generateQtable(qTable);
	}

	for (int i = 0; i < NUMBER_OF_TRIALS; i++)
	{
		for (int j = 0; j < NUMBER_OF_EPISODES; j++)
		{
			printf("[%d][%d] : %d\n",i,j,allSteps[i][j]);
		}
	}

	/*for (int trial = 0; trial < NUMBER_OF_TRIALS; trial++)
	{
		for (int i = 0; i < NUMBER_OF_EPISODES; i++)
		{
			printf("number of steps at [%d][%d] : %d\n", trial, i, allSteps[trial][i]);
		}
	}*/
	free(allSteps);
	free(qTable);
}