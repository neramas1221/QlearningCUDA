#include <stdio.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>


#include <time.h>

#define NUMBER_OF_STATES 11
#define NUMBER_OF_ACTIONS 4


void seedRandom()
{
	srand(time(NULL));
}
int transferFunction(int state, int action)
{
	int newState = 0;
	newState = state;

	switch (action)
	{
	case 0: 
		if (state == 0 || state == 1 || state == 2 || state == 3)
		{
			newState = state + 3;
			break;
		}
		else if (state == 4)
		{
			newState = state + 4;
			break;
		}
		else if (state == 5)
		{
			newState = state + 5; 
			break;
		}
	case 1:
		if (state == 6 || state == 7 || state == 8 || state == 9)
		{
			newState = state + 1;
			break;
		}
	case 2:
		if (state == 6 || state == 3 || state == 5 || state == 4)
		{
			newState = state - 3;
			break;
		}
	case 3:
		if (state == 10 || state == 7 || state == 8 || state == 9)
		{
			newState = state - 1;
			break;
		}
	}
	return newState;
}
void generateQtable(double **qTable)
{
	for (int i = 0; i < NUMBER_OF_STATES; i++)
	{
		for (int j = 0; j < NUMBER_OF_ACTIONS; j++)
		{
			qTable[i][j] = rand();
		}
	}
}
int getAction(double **qTable, int state)
{
	int action;
	double maxValue = -1;

	for (int i = 0; i < NUMBER_OF_ACTIONS; i++)
	{
		if (qTable[state][i] >maxValue)
		{
			maxValue = qTable[state][i];
				action = i;
		}
	}

	return action;
}
int getRndAction(double **qTable, int state)
{
	int action = 0;
	int maxValue = -1;
	// rnd number betwene 1-10
	for (int i = 0; i < NUMBER_OF_ACTIONS; i++)
	{
		if (qTable[state][i] >maxValue)
		{
			maxValue = qTable[state][i];
			action = i;
		}
	}
	return action;
}
int getReward(int state, int action)
{
	int reward = 0;
	if (state == 4 && action == 1)
	{
		reward = 10;
	}
	return reward;
}
double updateQTable(int state, int action, int nextState,int reward,double** qTable )
{
	double updateValue = 0;
	int nextAction;
	double discount = 0.9;
	double learningRate;
	double diff;

	nextAction = getAction(qTable, nextState);
	diff = reward + discount * qTable[nextState][nextAction] - qTable[state][action];
	updateValue = qTable[state][action] + learningRate * diff;

	return updateValue;
}
int main()
{

}