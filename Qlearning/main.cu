#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include "win-gettimeofday.h"


#define NUMBER_OF_STATES 11
#define NUMBER_OF_ACTIONS 4
#define NUMBER_OF_TRIALS 50
#define NUMBER_OF_EPISODES 200

void seedRandom()
{
	srand(time(NULL));
}
int transferFunction(int state, int action)
{
	int newState = 0;
	newState = state;

	switch (action)
	{
	case 0: 
		if (state == 0 || state == 1 || state == 2 || state == 3)
		{
			newState = state + 3;
			break;
		}
		else if (state == 4)
		{
			newState = state + 4;
			break;
		}
		else if (state == 5)
		{
			newState = state + 5; 
			break;
		}
	case 1:
		if (state == 6 || state == 7 || state == 8 || state == 9)
		{
			newState = state + 1;
			break;
		}
	case 2:
		if (state == 3 || state == 4 || state == 5 || state == 6)
		{
			newState = state - 3;
			break;
		}
		else if (state == 8)
		{
			newState = 4;
			break;
		}
		else if (state == 10)
		{
			newState = 5;
			break;
		}
	case 3:
		if (state == 10 || state == 7 || state == 8 || state == 9)
		{
			newState = state - 1;
			break;
		}
	}
	return newState;
}
void generateQtable(float **qTable)
{
	for (int i = 0; i < NUMBER_OF_STATES; i++)
	{
		for (int j = 0; j < NUMBER_OF_ACTIONS; j++)
		{
			qTable[i][j] = (double)rand() / (double)((unsigned)RAND_MAX + 1)* (0.1);
		}
	}
}
int getAction(float **qTable, int state)
{
	int action;
	double maxValue = -1;

	for (int i = 0; i < NUMBER_OF_ACTIONS; i++)
	{
		if (qTable[state][i] >maxValue)
		{
			maxValue = qTable[state][i];
				action = i;
		}
	}

	return action;
}
int getRndAction(float **qTable, int state)
{
	int action = 0;
	double maxValue = -1;
	int rndValue;
	rndValue = rand() % (10 + 1 - 1) + 1;
	if (rndValue <= 9)
	{
		for (int i = 0; i < NUMBER_OF_ACTIONS; i++)
		{
			if (qTable[state][i] > maxValue)
			{
				action = i;
				maxValue = qTable[state][i];
			}
		}
	}
	else
	{
		rndValue = rand() % (3 + 1 - 0) + 0;
		action = rndValue;
	}
	return action;
}
int getReward(int state, int action)
{
	int reward = 0;
	if (state == 4 && action == 2)
	{
		reward = 10;
	}
	return reward;
}
double updateQTable(int state, int action, int nextState,int reward, float** qTable )
{
	double updateValue = 0;
	int nextAction;
	double discount = 0.9;
	double learningRate =0.2;
	double diff;

	nextAction = getAction(qTable, nextState);
	diff = reward + discount * qTable[nextState][nextAction] - qTable[state][action];
	updateValue = qTable[state][action] + learningRate * diff;

	return updateValue;
}
int rndState()
{
	int state = 0;
	int Max = 10;
	state = rand() % Max;
	while (state == 1)
	{
		state = rand() % Max;
	}
	
	return state;
}
__device__ float calculateSTD(int episodeCounter[],int stepTotal,int size)
{
	float std;
	float mean = stepTotal / size;
	for (int i = 0; i < size; i++)
	{
		std += pow(episodeCounter[i] - mean, 2);
	}
	std = sqrt(std / size);
	return std;
}
__global__ void calculateAllSteps(int *StepsArray)
{
	__shared__ int episodeStorage[NUMBER_OF_EPISODES];
	__shared__ int trialStorage[NUMBER_OF_TRIALS];
 	__shared__ int total;
	__shared__ int *allSteps;
	allSteps = StepsArray;
	__shared__ int episodeTotal;
	__shared__ float std[NUMBER_OF_TRIALS];
	__shared__ int worstSteps;
	__shared__ int bestSteps;
	__shared__ int totalSteps;
	 bestSteps = 100000;
	 worstSteps = 0;
	total = 0; 
	episodeTotal = 0;
	totalSteps = 0;
	__syncthreads();
	for (int i = 0; i < NUMBER_OF_TRIALS;i++)
	{
		for (int j = 0; j < NUMBER_OF_EPISODES; j+=20)
		{
			/* gets the element from the array using the stride values for example if i is 2 and j is 1 then the index
			 would be 201 meaning it is the second episode of the second trial;*/
			total += allSteps[i*NUMBER_OF_EPISODES + j];
			total += allSteps[i*NUMBER_OF_EPISODES + j+1];
			total += allSteps[i*NUMBER_OF_EPISODES + j+2];
			total += allSteps[i*NUMBER_OF_EPISODES + j+3];
			total += allSteps[i*NUMBER_OF_EPISODES + j+4];
			total += allSteps[i*NUMBER_OF_EPISODES + j+5];
			total += allSteps[i*NUMBER_OF_EPISODES + j+6];
			total += allSteps[i*NUMBER_OF_EPISODES + j+7];
			total += allSteps[i*NUMBER_OF_EPISODES + j+8];
			total += allSteps[i*NUMBER_OF_EPISODES + j+9];
			total += allSteps[i*NUMBER_OF_EPISODES + j+10];
			total += allSteps[i*NUMBER_OF_EPISODES + j+11];
			total += allSteps[i*NUMBER_OF_EPISODES + j+12];
			total += allSteps[i*NUMBER_OF_EPISODES + j+13];
			total += allSteps[i*NUMBER_OF_EPISODES + j+14];
			total += allSteps[i*NUMBER_OF_EPISODES + j+15];
			total += allSteps[i*NUMBER_OF_EPISODES + j+16];
			total += allSteps[i*NUMBER_OF_EPISODES + j+17];
			total += allSteps[i*NUMBER_OF_EPISODES + j+18];
			total += allSteps[i*NUMBER_OF_EPISODES + j+19];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+1];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+2];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+3];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+4];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+5];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+6];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+7];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+8];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+9];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+10];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+11];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+12];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+13];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+14];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+15];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+16];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+17];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+18];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES + j+19];
		}
		std[i] = calculateSTD(episodeStorage,total,NUMBER_OF_EPISODES);
		trialStorage[i] = total;
		totalSteps += total;
		episodeTotal += (total / 200);
		total = 0;
	}

	float aveStd = 0;
	float mean = totalSteps / NUMBER_OF_TRIALS;
	for (int i = 0; i < NUMBER_OF_TRIALS; i++)
	{
		aveStd += pow(std[i] - mean, 2);
	}
	aveStd = sqrt(aveStd / NUMBER_OF_TRIALS);

	for (int i = 0; i < NUMBER_OF_EPISODES*NUMBER_OF_TRIALS; i++)
	{
		if (bestSteps > allSteps[i])
		{
			bestSteps = allSteps[i];
		}
		if (worstSteps < allSteps[i])
		{
			worstSteps = allSteps[i];
		}
		__syncthreads();
	}

	__syncthreads();
	int block_id = blockIdx.x + gridDim.x * blockIdx.y;
	unsigned int id = blockDim.x * block_id + threadIdx.x;
	//printf("id : %d\n", id);
	if (id == 1)
	{
		printf("Data from trials \n\n");
		printf("avarge Standard deviation : %f\n", aveStd);
		printf("Best value %d\n", bestSteps);
		printf("worst value %d\n", worstSteps);
		printf("Avarage steps per tiral : %d\n",episodeTotal);
		printf("Avarage steps per episode : %d\n", episodeTotal/50);
		printf("\n\n");
	}
}
int main()
{
	long long totalProgrammeTimmer = start_timer();
	seedRandom();
	int size = sizeof(int) * NUMBER_OF_EPISODES * NUMBER_OF_TRIALS;
	int returnSize = sizeof(int) * 4;
	float **qTable;
	qTable = (float**)  malloc(NUMBER_OF_STATES * sizeof(float*));
	for (int i = 0; i < NUMBER_OF_STATES; i++)
	{
		qTable[i] = (float*)malloc(sizeof(float)*NUMBER_OF_ACTIONS);
	}
	generateQtable(qTable);
		
	int *allSteps;
	allSteps = (int*)malloc((NUMBER_OF_TRIALS * NUMBER_OF_EPISODES)* sizeof(int));
	/*allSteps[0] = (int*)malloc(NUMBER_OF_EPISODES * sizeof(int));
	for (size_t i = 1; i < NUMBER_OF_TRIALS; i++)
	{
		allSteps[i] = allSteps[i - 1] + NUMBER_OF_EPISODES;
	}*/
	/*for (int i = 0; i < NUMBER_OF_TRIALS; i++)
	{
		allSteps[i] = (int*)malloc(sizeof(int)* NUMBER_OF_EPISODES);
	}*/
	

	int state = 0;
	int action = 0;
	int newState = 0;
	int reward = 0;
	int steps = 0;
	float updatedValue = 0;

	for (int trial = 0; trial < NUMBER_OF_TRIALS; trial++)
	{
		for (int i = 0; i < NUMBER_OF_EPISODES; i++)
		{
			state = rndState();
			while (state != 1)
			{
				action = getRndAction(qTable, state);
				reward = getReward(state, action);
				newState = transferFunction(state, action);
				updatedValue = updateQTable(state, action, newState, reward, qTable);
				qTable[state][action] = updatedValue;
				
				state = newState;
				steps++;
			}
			
			allSteps[trial*NUMBER_OF_EPISODES + i] = steps;
			
			steps = 0;
			
		}
		generateQtable(qTable);
	}


	/*int *returnData;
	returnData =(int*) malloc(4 * sizeof(int));*/
	int *d_allsteps;
	//int *d_returnData;
	long long transferTimmer = start_timer();

	hipStream_t stream1;
	hipError_t err;
	err = hipStreamCreate(&stream1);

	printf("CUDA Stream: %s\n", hipGetErrorString(err));

	err = hipMalloc((void**)&d_allsteps, size);
	
	printf("CUDA malloc 1D array: %s\n", hipGetErrorString(err));

	//err = hipMalloc((void**)&d_returnData, returnSize);

	//printf("CUDA malloc 1D array: %s\n", hipGetErrorString(err));

	err = hipMemcpyAsync(d_allsteps, allSteps,size, hipMemcpyHostToDevice,stream1);

	printf("CUDA memcpy 1D array: %s\n", hipGetErrorString(err));

	//err = hipMemset(d_returnData, 0, returnSize); //(d_returnData,returnData,returnSize,hipMemcpyHostToDevice);

	//printf("CUDA memcpy 1D array of nothing: %s\n", hipGetErrorString(err));

	stop_timer(transferTimmer,"Transfer timer");

	long long GPUComput = start_timer();
	int gridSize = (int)ceil(NUMBER_OF_EPISODES*NUMBER_OF_TRIALS / 50);
	dim3 dimGrid(gridSize,1,1);
	dim3 dimBlock(50,1,1);

	calculateAllSteps << <gridSize, 50,0 ,stream1 >> > (d_allsteps);

	//err = hipMemcpy(returnData, d_returnData, returnSize, hipMemcpyDeviceToHost);

	//printf("CUDA memcpy data back: %s\n", hipGetErrorString(err));
	stop_timer(GPUComput, "Computaion time");
	/*printf("Best value %d\n", returnData[2]);
	printf("worst value %d\n", returnData[3]);
	printf("Avarage steps per tiral : %d\n", returnData[0]);
	printf("Avarage steps per episode : %d\n", returnData[1]);*/
	stop_timer(totalProgrammeTimmer, "Total programme time");

	hipFree(d_allsteps);

	//hipFree(d_returnData);

	free(allSteps);

	free(qTable);
}