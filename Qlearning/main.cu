#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include "win-gettimeofday.h"


#define NUMBER_OF_STATES 11
#define NUMBER_OF_ACTIONS 4
#define NUMBER_OF_TRIALS 50
#define NUMBER_OF_EPISODES 200

void seedRandom()
{
	srand(time(NULL));
}
int transferFunction(int state, int action)
{
	int newState = 0;
	newState = state;

	switch (action)
	{
	case 0: 
		if (state == 0 || state == 1 || state == 2 || state == 3)
		{
			newState = state + 3;
			break;
		}
		else if (state == 4)
		{
			newState = state + 4;
			break;
		}
		else if (state == 5)
		{
			newState = state + 5; 
			break;
		}
	case 1:
		if (state == 6 || state == 7 || state == 8 || state == 9)
		{
			newState = state + 1;
			break;
		}
	case 2:
		if (state == 3 || state == 4 || state == 5 || state == 6)
		{
			newState = state - 3;
			break;
		}
		else if (state == 8)
		{
			newState = 4;
			break;
		}
		else if (state == 10)
		{
			newState = 5;
			break;
		}
	case 3:
		if (state == 10 || state == 7 || state == 8 || state == 9)
		{
			newState = state - 1;
			break;
		}
	}
	return newState;
}
void generateQtable(double **qTable)
{
	for (int i = 0; i < NUMBER_OF_STATES; i++)
	{
		for (int j = 0; j < NUMBER_OF_ACTIONS; j++)
		{
			qTable[i][j] = (double)rand() / (double)((unsigned)RAND_MAX + 1)* (0.1);
		}
	}
}
int getAction(double **qTable, int state)
{
	int action;
	double maxValue = -1;

	for (int i = 0; i < NUMBER_OF_ACTIONS; i++)
	{
		if (qTable[state][i] >maxValue)
		{
			maxValue = qTable[state][i];
				action = i;
		}
	}

	return action;
}
int getRndAction(double **qTable, int state)
{
	int action = 0;
	double maxValue = -1;
	int rndValue;
	rndValue = rand() % (10 + 1 - 1) + 1;
	if (rndValue <= 9)
	{
		for (int i = 0; i < NUMBER_OF_ACTIONS; i++)
		{
			if (qTable[state][i] > maxValue)
			{
				action = i;
				maxValue = qTable[state][i];
			}
		}
	}
	else
	{
		rndValue = rand() % (3 + 1 - 0) + 0;
		action = rndValue;
	}
	return action;
}
int getReward(int state, int action)
{
	int reward = 0;
	if (state == 4 && action == 2)
	{
		reward = 10;
	}
	return reward;
}
double updateQTable(int state, int action, int nextState,int reward,double** qTable )
{
	double updateValue = 0;
	int nextAction;
	double discount = 0.9;
	double learningRate =0.2;
	double diff;

	nextAction = getAction(qTable, nextState);
	diff = reward + discount * qTable[nextState][nextAction] - qTable[state][action];
	updateValue = qTable[state][action] + learningRate * diff;

	return updateValue;
}
int rndState()
{
	int state = 0;
	int Max = 10;
	state = rand() % Max;
	while (state == 1)
	{
		state = rand() % Max;
	}
	
	return state;
}
__device__ double calculateSTD(int episodeCounter[],int stepTotal)
{
	double std;
	double mean = stepTotal / NUMBER_OF_EPISODES;
	for (int i = 0; i < NUMBER_OF_EPISODES; i++)
	{
		std += pow(episodeCounter[i] - mean, 2);
	}
	std = sqrt(std / NUMBER_OF_EPISODES);
	return std;
}
__global__ void calculateAllSteps(int *StepsArray,int* returnData)
{
	__shared__ int episodeStorage[NUMBER_OF_EPISODES];
	__shared__ int trialStorage[NUMBER_OF_TRIALS];
 	__shared__ int total;
	__shared__ int *allSteps;
	allSteps = StepsArray;
	__shared__ int episodeTotal;
	//double std[NUMBER_OF_TRIALS];
	__shared__ int worstSteps;
	__shared__ int bestSteps;
	 bestSteps = 100000;
	 worstSteps = 0;
	total = 0; 
	episodeTotal = 0;
	__syncthreads();
	for (int i = 0; i < NUMBER_OF_TRIALS;i++)
	{
		for (int j = 0; j < NUMBER_OF_EPISODES; j++)
		{
			/* gets the element from the array using the stride values for example if i is 2 and j is 1 then the index
			 would be 201 meaning it is the second episode of the second trial;*/
			total += allSteps[i*NUMBER_OF_EPISODES +j];
			episodeStorage[j] = allSteps[i*NUMBER_OF_EPISODES +j];
		
		}
		//std[i] = calculateSTD(episodeStorage,total);
		/*printf("Best value %d\n",bestValues);
		printf("worst value %d\n", worstValue);*/ 
		trialStorage[i] = total;
		episodeTotal += (total / 200);
		total = 0;
	}

	for (int i = 0; i < NUMBER_OF_EPISODES*NUMBER_OF_TRIALS; i++)
	{
		if (bestSteps > allSteps[i])
		{
			bestSteps = allSteps[i];
		}
		if (worstSteps < allSteps[i])
		{
			worstSteps = allSteps[i];
		}
		__syncthreads();
	}

	__syncthreads();
	returnData[0] = episodeTotal;
	returnData[1] = episodeTotal / 50;
	returnData[2] = bestSteps;
	returnData[3] = worstSteps;
}
int main()
{
	long long totalProgrammeTimmer = start_timer();
	seedRandom();
	int size = sizeof(int) * NUMBER_OF_EPISODES * NUMBER_OF_TRIALS;
	int returnSize = sizeof(int) * 4;
	double **qTable;
	qTable = (double**)  malloc(NUMBER_OF_STATES * sizeof(double*));
	for (int i = 0; i < NUMBER_OF_STATES; i++)
	{
		qTable[i] = (double*)malloc(sizeof(double)*NUMBER_OF_ACTIONS);
	}
	generateQtable(qTable);
		
	int *allSteps;
	allSteps = (int*)malloc((NUMBER_OF_TRIALS * NUMBER_OF_EPISODES)* sizeof(int));
	/*allSteps[0] = (int*)malloc(NUMBER_OF_EPISODES * sizeof(int));
	for (size_t i = 1; i < NUMBER_OF_TRIALS; i++)
	{
		allSteps[i] = allSteps[i - 1] + NUMBER_OF_EPISODES;
	}*/
	/*for (int i = 0; i < NUMBER_OF_TRIALS; i++)
	{
		allSteps[i] = (int*)malloc(sizeof(int)* NUMBER_OF_EPISODES);
	}*/
	

	int state = 0;
	int action = 0;
	int newState = 0;
	int reward = 0;
	int steps = 0;
	double updatedValue = 0;

	for (int trial = 0; trial < NUMBER_OF_TRIALS; trial++)
	{
		for (int i = 0; i < NUMBER_OF_EPISODES; i++)
		{
			state = rndState();
			while (state != 1)
			{
				action = getRndAction(qTable, state);
				reward = getReward(state, action);
				newState = transferFunction(state, action);
				updatedValue = updateQTable(state, action, newState, reward, qTable);
				qTable[state][action] = updatedValue;
				
				state = newState;
				steps++;
			}
			
			allSteps[trial*NUMBER_OF_EPISODES + i] = steps;
			
			steps = 0;
			
		}
		generateQtable(qTable);
	}


	int *returnData;
	returnData =(int*) malloc(4 * sizeof(int));
	int *d_allsteps;
	int *d_returnData;
	int avargeTrialSteps, avrageEpisodeSteps, bestSteps,worstSteps;

	long long transferTimmer = start_timer();

	hipError_t err = hipMalloc((void**)&d_allsteps, size);

	printf("CUDA malloc 1D array: %s\n", hipGetErrorString(err));

	err = hipMalloc((void**)&d_returnData, returnSize);

	printf("CUDA malloc 1D array: %s\n", hipGetErrorString(err));

	err = hipMemcpy(d_allsteps, allSteps,size, hipMemcpyHostToDevice);

	printf("CUDA memcpy 1D array: %s\n", hipGetErrorString(err));

	err = hipMemcpy(d_returnData,returnData,returnSize,hipMemcpyHostToDevice);

	printf("CUDA memcpy 1D array of nothing: %s\n", hipGetErrorString(err));

	stop_timer(transferTimmer,"Transfer timer");

	long long GPUComput = start_timer();
	int gridSize = (int)ceil(NUMBER_OF_EPISODES*NUMBER_OF_TRIALS / 16);
	dim3 dimGrid(gridSize,1,1);
	dim3 dimBlock(1,1,1);

	calculateAllSteps << <gridSize,1>> > (d_allsteps, d_returnData);

	err = hipMemcpy(returnData, d_returnData, returnSize, hipMemcpyDeviceToHost);

	printf("CUDA memcpy data back: %s\n", hipGetErrorString(err));
	stop_timer(GPUComput, "Computaion time");
	printf("Best value %d\n", returnData[2]);
	printf("worst value %d\n", returnData[3]);
	printf("Avarage steps per tiral : %d\n", returnData[0]);
	printf("Avarage steps per episode : %d\n", returnData[1]);
	stop_timer(totalProgrammeTimmer, "Total programme time");

	hipFree(d_allsteps);

	hipFree(d_returnData);

	free(allSteps);

	free(qTable);
}