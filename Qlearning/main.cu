#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>

#define NUMBER_OF_STATES 11
#define NUMBER_OF_ACTIONS 4
#define NUMBER_OF_TRIALS 50
#define NUMBER_OF_EPISODES 20

void seedRandom()
{
	srand(time(NULL));
}
int transferFunction(int state, int action)
{
	int newState = 0;
	newState = state;

	switch (action)
	{
	case 0: 
		if (state == 0 || state == 1 || state == 2 || state == 3)
		{
			newState = state + 3;
			break;
		}
		else if (state == 4)
		{
			newState = state + 4;
			break;
		}
		else if (state == 5)
		{
			newState = state + 5; 
			break;
		}
	case 1:
		if (state == 6 || state == 7 || state == 8 || state == 9)
		{
			newState = state + 1;
			break;
		}
	case 2:
		if (state == 3 || state == 4 || state == 5 || state == 6)
		{
			newState = state - 3;
			break;
		}
		else if (state == 8)
		{
			newState = 4;
			break;
		}
		else if (state == 10)
		{
			newState = 5;
			break;
		}
	case 3:
		if (state == 10 || state == 7 || state == 8 || state == 9)
		{
			newState = state - 1;
			break;
		}
	}
	return newState;
}
void generateQtable(double **qTable)
{
	for (int i = 0; i < NUMBER_OF_STATES; i++)
	{
		for (int j = 0; j < NUMBER_OF_ACTIONS; j++)
		{
			qTable[i][j] = (double)rand() / (double)((unsigned)RAND_MAX + 1)* (0.1);
		}
	}
}
int getAction(double **qTable, int state)
{
	int action;
	double maxValue = -1;

	for (int i = 0; i < NUMBER_OF_ACTIONS; i++)
	{
		if (qTable[state][i] >maxValue)
		{
			maxValue = qTable[state][i];
				action = i;
		}
	}

	return action;
}
int getRndAction(double **qTable, int state)
{
	int action = 0;
	double maxValue = -1;
	int rndValue;
	rndValue = rand() % (10 + 1 - 1) + 1;
	if (rndValue <= 9)
	{
		for (int i = 0; i < NUMBER_OF_ACTIONS; i++)
		{
			if (qTable[state][i] > maxValue)
			{
				action = i;
				maxValue = qTable[state][i];
			}
		}
	}
	else
	{
		rndValue = rand() % (3 + 1 - 0) + 0;
		action = rndValue;
	}
	return action;
}
int getReward(int state, int action)
{
	int reward = 0;
	if (state == 4 && action == 2)
	{
		reward = 10;
	}
	return reward;
}
double updateQTable(int state, int action, int nextState,int reward,double** qTable )
{
	double updateValue = 0;
	int nextAction;
	double discount = 0.9;
	double learningRate =0.2;
	double diff;

	nextAction = getAction(qTable, nextState);
	diff = reward + discount * qTable[nextState][nextAction] - qTable[state][action];
	updateValue = qTable[state][action] + learningRate * diff;

	return updateValue;
}
int rndState()
{
	int state = 0;
	int Max = 10;
	state = rand() % Max;
	while (state == 1)
	{
		state = rand() % Max;
	}
	
	return state;
}
__device__ double calculateSTD(int episodeCounter[],int stepTotal)
{
	double std;
	double mean = stepTotal / NUMBER_OF_EPISODES;
	for (int i = 0; i < NUMBER_OF_EPISODES; i++)
	{
		std += pow(episodeCounter[i] - mean, 2);
	}
	std = sqrt(std / NUMBER_OF_EPISODES);
	return std;
}
__global__ void calculateAllSteps(int *StepsArray)
{
	__shared__ int epsiodeCounter[NUMBER_OF_EPISODES];
	__shared__ int trialCounter[NUMBER_OF_TRIALS];
	__shared__ int total;
	double std[NUMBER_OF_TRIALS];
	total = 0; 
	for (int i = 0; i < NUMBER_OF_TRIALS;i++)
	{
		for (int j = 0; j < NUMBER_OF_EPISODES; j++)
		{
			// gets the element from the array using the stride values for example if i is 2 and j is 1 then the index
			// would be 201 meaning it is the second episode of the second trial;
			total += StepsArray[i*200+j];
			epsiodeCounter[j] = StepsArray[i];
		}
		std[i] = calculateSTD(epsiodeCounter,total);
		trialCounter[i] = total;
		total = 0;
	}
	printf("STD : %f", std[0]);
	
}
int main()
{
	seedRandom();
	int size = sizeof(int) * NUMBER_OF_EPISODES * NUMBER_OF_TRIALS;
	double **qTable;
	qTable = (double**)  malloc(NUMBER_OF_STATES * sizeof(double*));
	for (int i = 0; i < NUMBER_OF_STATES; i++)
	{
		qTable[i] = (double*)malloc(sizeof(double)*NUMBER_OF_ACTIONS);
	}
	generateQtable(qTable);
		/*for (int i = 0; i < NUMBER_OF_STATES; i++)
		{
			for (int j = 0; j < NUMBER_OF_ACTIONS; j++)
			{
				printf("vlaue at [%d][%d] : %f\n", i, j, qTable[i][j]);
			}
		}*/
	int *allSteps;
	allSteps = (int*)malloc((NUMBER_OF_TRIALS * NUMBER_OF_EPISODES)* sizeof(int));
	/*allSteps[0] = (int*)malloc(NUMBER_OF_EPISODES * sizeof(int));
	for (size_t i = 1; i < NUMBER_OF_TRIALS; i++)
	{
		allSteps[i] = allSteps[i - 1] + NUMBER_OF_EPISODES;
	}*/
	/*for (int i = 0; i < NUMBER_OF_TRIALS; i++)
	{
		allSteps[i] = (int*)malloc(sizeof(int)* NUMBER_OF_EPISODES);
	}*/
	

	int state = 0;
	int action = 0;
	int newState = 0;
	int reward = 0;
	int steps = 0;
	double updatedValue = 0;

	for (int trial = 0; trial < NUMBER_OF_TRIALS; trial++)
	{
		for (int i = 0; i < NUMBER_OF_EPISODES; i++)
		{
			state = rndState();
			while (state != 1)
			{
				action = getRndAction(qTable, state);
				reward = getReward(state, action);
				newState = transferFunction(state, action);
				updatedValue = updateQTable(state, action, newState, reward, qTable);
				qTable[state][action] = updatedValue;
				//printf("state : %d , action : %d newState : %d\n", state, action, newState);
				state = newState;
				steps++;
			}
			
			allSteps[trial*NUMBER_OF_EPISODES + i] = steps;
			
			steps = 0;
			
		}
		generateQtable(qTable);
	}
	int *d_allsteps;
	
	hipError_t err = hipMalloc((void**)&d_allsteps, size);
	printf("CUDA malloc 1D array: %s\n", hipGetErrorString(err));
	err = hipMemcpy(d_allsteps, allSteps,size, hipMemcpyHostToDevice);
	printf("CUDA memcpy 1D array: %s\n", hipGetErrorString(err));
	calculateAllSteps << <1,1>> > (d_allsteps);
	hipFree(d_allsteps);
	free(allSteps);
	free(qTable);
}